#include "hip/hip_runtime.h"
#include "denoise.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

const int KERNEL_DIM = 3;

__device__ bool pixel_cmp(uchar3& a, uchar3& b)
{
    return (a.x + a.y + a.z > b.x + b.y + b.z);
}

__device__ uchar3 median(uchar3 vector[], int count)
{
    // bubble sort
    for (int i = 0; i < count; ++i)
    {
        for (int j = 0; j < count - i; ++j)
        {
            if (pixel_cmp(vector[j], vector[j + 1]))
            {
                uchar3 temp = vector[j];
                vector[j] = vector[j + 1];
                vector[j + 1] = temp;
            }
        }
    }

    // find median value
    int index = count / 2;
    /*if (count % 2 == 0)
    {
        uchar3 median_value;
        median_value.x = (vector[index].x + vector[index + 1].x) / 2;
        median_value.y = (vector[index].y + vector[index + 1].y) / 2;
        median_value.z = (vector[index].z + vector[index + 1].z) / 2;
        return median_value;
    }*/
    return vector[index];
}

__global__ void denoise_kernel(uchar3* input_image, uchar3* output_image, int width, int height)
{
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;

    if (offset < width * height)
    {
        float r = 0, g = 0, b = 0;
        int count = 0;
        uchar3 vector[KERNEL_DIM * KERNEL_DIM];

        for (int ox = -(KERNEL_DIM / 2); ox < (KERNEL_DIM / 2) + 1; ox++)
        {
            for (int oy = -(KERNEL_DIM / 2); oy < (KERNEL_DIM / 2) + 1; oy++)
            {
                if ((x + ox) > -1 && (x + ox) < width && (y + oy) > -1 && (y + oy) < height)
                {
                    const int current_offset = offset + ox + oy * width;

                    vector[count].x = input_image[current_offset].x;
                    vector[count].y = input_image[current_offset].y;
                    vector[count].z = input_image[current_offset].z;

                    ++count;
                }
            }
        }
        output_image[offset] = median(vector, count);
    }
}

void getError(hipError_t err)
{
    if (err != hipSuccess)
    {
        std::cout << "Error " << hipGetErrorString(err) << std::endl;
    }
}

void denoise(unsigned char* input_image, unsigned char* output_image, int width, int height)
{
    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc((void**)&dev_input, width * height * 3 * sizeof(unsigned char)));
    getError(hipMemcpy(dev_input, input_image, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    getError(hipMalloc((void**)&dev_output, width * height * 3 * sizeof(unsigned char)));

    dim3 blockDims(512, 1, 1);
    dim3 gridDims(ceil(double(width * height) / blockDims.x), 1, 1);

    denoise_kernel<<<gridDims, blockDims>>>((uchar3*)dev_input, (uchar3*)dev_output, width, height);

    getError(hipMemcpy(output_image, dev_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));
}
